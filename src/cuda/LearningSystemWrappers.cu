#include "hip/hip_runtime.h"
// ============================================================================
// CUDA WRAPPER FUNCTIONS IMPLEMENTATION
// File: src/cuda/LearningSystemWrappers.cu
// ============================================================================

#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <cstdio>

// Include GPU structures (must be available in .cu compilation)
#include <NeuroGen/cuda/GPUNeuralStructures.h>

// ============================================================================
// CUDA KERNEL IMPLEMENTATIONS
// ============================================================================

/**
 * @brief Eligibility trace reset kernel with biological decay dynamics
 */
__global__ void eligibility_trace_reset_kernel(GPUSynapse* synapses, int num_synapses) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    // Biological eligibility trace reset with protein degradation
    synapse.eligibility_trace *= 0.95f; // Rapid degradation
    synapse.protein_synthesis_rate *= 0.99f; // Slower protein degradation
    
    // Reset dopamine sensitivity if below threshold
    if (synapse.dopamine_sensitivity < 0.1f) {
        synapse.dopamine_sensitivity = 0.5f; // Baseline sensitivity
    }
}

/**
 * @brief Enhanced STDP kernel with multi-factor plasticity
 */
__global__ void enhanced_stdp_kernel(
    GPUSynapse* synapses,
    const GPUNeuronState* neurons,
    float current_time,
    float dt,
    int num_synapses
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    // Validate neuron indices
    if (synapse.pre_neuron_id >= 0 && synapse.post_neuron_id >= 0) {
        const GPUNeuronState& pre_neuron = neurons[synapse.pre_neuron_id];
        const GPUNeuronState& post_neuron = neurons[synapse.post_neuron_id];
        
        // Calculate spike timing difference
        float delta_t = post_neuron.last_spike_time - pre_neuron.last_spike_time;
        
        // Biological STDP with calcium-dependent modulation
        float stdp_window = 20.0f; // 20ms STDP window
        float calcium_factor = post_neuron.calcium_concentration / 1.0f; // Normalize
        
        if (fabsf(delta_t) < stdp_window) {
            float stdp_magnitude;
            
            if (delta_t > 0) {
                // LTP: Post before pre (causal)
                stdp_magnitude = __expf(-delta_t / 10.0f) * calcium_factor;
                synapse.weight += stdp_magnitude * 0.01f * dt;
            } else {
                // LTD: Pre before post (anti-causal)
                stdp_magnitude = __expf(delta_t / 10.0f) * calcium_factor;
                synapse.weight -= stdp_magnitude * 0.005f * dt;
            }
            
            // Update eligibility trace
            synapse.eligibility_trace += stdp_magnitude * 0.1f;
        }
        
        // Bound synaptic weight
        synapse.weight = fmaxf(0.0f, fminf(synapse.weight, 5.0f));
    }
}

/**
 * @brief Eligibility trace update kernel with biological dynamics
 */
__global__ void eligibility_trace_update_kernel(
    GPUSynapse* synapses,
    const GPUNeuronState* neurons,
    float current_time,
    float dt,
    int num_synapses
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    // Exponential decay of eligibility traces
    float decay_rate = 1.0f / 1000.0f; // 1 second time constant
    float decay_factor = __expf(-dt * decay_rate);
    
    synapse.eligibility_trace *= decay_factor;
    
    // Update based on recent synaptic activity
    if (synapse.pre_neuron_id >= 0 && synapse.post_neuron_id >= 0) {
        const GPUNeuronState& pre_neuron = neurons[synapse.pre_neuron_id];
        const GPUNeuronState& post_neuron = neurons[synapse.post_neuron_id];
        
        // Add contribution from recent activity
        float activity_contribution = pre_neuron.membrane_potential * post_neuron.membrane_potential;
        synapse.eligibility_trace += activity_contribution * 0.001f * dt;
    }
    
    // Bound eligibility trace
    synapse.eligibility_trace = fmaxf(0.0f, fminf(synapse.eligibility_trace, 2.0f));
}

/**
 * @brief Trace monitoring kernel for statistics collection
 */
__global__ void trace_monitoring_kernel(
    const GPUSynapse* synapses,
    int num_synapses,
    float* trace_stats
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    // Use shared memory for efficient reduction
    __shared__ float shared_sum[256];
    __shared__ float shared_max[256];
    
    int tid = threadIdx.x;
    shared_sum[tid] = 0.0f;
    shared_max[tid] = 0.0f;
    
    // Process multiple synapses per thread if necessary
    if (idx < num_synapses) {
        float trace_value = synapses[idx].eligibility_trace;
        shared_sum[tid] = trace_value;
        shared_max[tid] = trace_value;
    }
    
    __syncthreads();
    
    // Reduction for statistics
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid + stride]);
        }
        __syncthreads();
    }
    
    // Write results to global memory
    if (tid == 0) {
        atomicAdd(&trace_stats[0], shared_sum[0]); // Sum
        atomicMax((int*)&trace_stats[1], __float_as_int(shared_max[0])); // Max
    }
}

/**
 * @brief Reward modulation kernel with dopaminergic dynamics
 */
__global__ void reward_modulation_kernel(
    GPUSynapse* synapses,
    GPUNeuronState* neurons,
    float reward,
    float current_time,
    float dt,
    int num_synapses
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    // Dopaminergic modulation with realistic dynamics
    float dopamine_release = reward * (1.0f + 0.1f * sinf(current_time * 0.01f));
    float dopamine_decay = __expf(-dt / 100.0f); // 100ms dopamine half-life
    
    // Update dopamine concentration
    synapse.dopamine_level = synapse.dopamine_level * dopamine_decay + dopamine_release * dt;
    
    // Apply reward-dependent plasticity
    float eligibility_weighted_change = synapse.eligibility_trace * synapse.dopamine_level;
    float learning_rate = 0.001f * synapse.dopamine_sensitivity;
    
    synapse.weight += eligibility_weighted_change * learning_rate * dt;
    
    // Update dopamine sensitivity (metaplasticity)
    synapse.dopamine_sensitivity += (dopamine_release - 0.5f) * 0.0001f * dt;
    synapse.dopamine_sensitivity = fmaxf(0.1f, fminf(synapse.dopamine_sensitivity, 2.0f));
    
    // Bound synaptic weight
    synapse.weight = fmaxf(0.0f, fminf(synapse.weight, 10.0f));
}

/**
 * @brief Hebbian learning kernel with BCM-like threshold adaptation
 */
__global__ void hebbian_learning_kernel(
    GPUSynapse* synapses,
    const GPUNeuronState* neurons,
    float current_time,
    float dt,
    int num_synapses
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    if (synapse.pre_neuron_id >= 0 && synapse.post_neuron_id >= 0) {
        const GPUNeuronState& pre_neuron = neurons[synapse.pre_neuron_id];
        const GPUNeuronState& post_neuron = neurons[synapse.post_neuron_id];
        
        // BCM-like plasticity with sliding threshold
        float pre_activity = pre_neuron.membrane_potential / 70.0f; // Normalize
        float post_activity = post_neuron.membrane_potential / 70.0f;
        
        // Sliding threshold based on recent activity
        float activity_threshold = post_neuron.average_activity;
        
        // BCM rule: LTP if above threshold, LTD if below
        float plasticity_signal = post_activity * (post_activity - activity_threshold) * pre_activity;
        
        // Apply Hebbian change
        float hebbian_rate = 0.0001f;
        synapse.weight += plasticity_signal * hebbian_rate * dt;
        
        // Bound weight
        synapse.weight = fmaxf(0.0f, fminf(synapse.weight, 5.0f));
    }
}

/**
 * @brief BCM learning kernel with explicit threshold dynamics
 */
__global__ void bcm_learning_kernel(
    GPUSynapse* synapses,
    const GPUNeuronState* neurons,
    float learning_rate,
    float dt,
    int num_synapses
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    if (synapse.pre_neuron_id >= 0 && synapse.post_neuron_id >= 0) {
        const GPUNeuronState& pre_neuron = neurons[synapse.pre_neuron_id];
        const GPUNeuronState& post_neuron = neurons[synapse.post_neuron_id];
        
        // BCM plasticity with homeostatic threshold
        float pre_rate = pre_neuron.firing_rate;
        float post_rate = post_neuron.firing_rate;
        float threshold = post_neuron.bcm_threshold;
        
        // BCM learning rule
        float weight_change = learning_rate * pre_rate * post_rate * (post_rate - threshold) * dt;
        
        synapse.weight += weight_change;
        synapse.weight = fmaxf(0.0f, fminf(synapse.weight, 5.0f));
    }
}

/**
 * @brief Correlation-based learning kernel with matrix operations
 */
__global__ void correlation_learning_kernel(
    GPUSynapse* synapses,
    const GPUNeuronState* neurons,
    float* correlation_matrix,
    float learning_rate,
    float dt,
    int num_synapses,
    int matrix_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_synapses) return;
    
    GPUSynapse& synapse = synapses[idx];
    
    if (synapse.pre_neuron_id >= 0 && synapse.post_neuron_id >= 0 && 
        synapse.pre_neuron_id < matrix_size && synapse.post_neuron_id < matrix_size) {
        
        const GPUNeuronState& pre_neuron = neurons[synapse.pre_neuron_id];
        const GPUNeuronState& post_neuron = neurons[synapse.post_neuron_id];
        
        // Update correlation matrix
        int matrix_idx = synapse.post_neuron_id * matrix_size + synapse.pre_neuron_id;
        float correlation = pre_neuron.membrane_potential * post_neuron.membrane_potential;
        
        // Exponential moving average of correlations
        correlation_matrix[matrix_idx] = correlation_matrix[matrix_idx] * 0.99f + correlation * 0.01f;
        
        // Apply correlation-based learning
        float correlation_strength = correlation_matrix[matrix_idx];
        synapse.weight += correlation_strength * learning_rate * dt;
        
        // Bound weight
        synapse.weight = fmaxf(0.0f, fminf(synapse.weight, 5.0f));
    }
}

/**
 * @brief Reward prediction error computation kernel
 */
__global__ void reward_prediction_error_kernel(
    const float* actual_reward,
    float* predicted_rewards,
    int num_timesteps
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_timesteps) return;
    
    // Simple TD error computation
    float prediction_error = *actual_reward - predicted_rewards[idx];
    
    // Update prediction using simple learning rule
    predicted_rewards[idx] += 0.1f * prediction_error;
    
    // Store error for further processing (could be expanded)
    // This is a simplified version - full implementation would include
    // temporal difference learning with value function approximation
}

// ============================================================================
// C++ WRAPPER FUNCTIONS (called from CPU code)
// ============================================================================

extern "C" {

void launch_eligibility_reset_wrapper(void* d_synapses, int num_synapses) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    eligibility_trace_reset_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses), num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in eligibility_reset: %s\n", hipGetErrorString(error));
    }
}

void launch_enhanced_stdp_wrapper(
    void* d_synapses, 
    const void* d_neurons,
    float current_time, 
    float dt, 
    int num_synapses
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    enhanced_stdp_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<const GPUNeuronState*>(d_neurons),
        current_time, dt, num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in enhanced_stdp: %s\n", hipGetErrorString(error));
    }
}

void launch_eligibility_update_wrapper(
    void* d_synapses,
    const void* d_neurons,
    float current_time,
    float dt,
    int num_synapses
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    eligibility_trace_update_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<const GPUNeuronState*>(d_neurons),
        current_time, dt, num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in eligibility_update: %s\n", hipGetErrorString(error));
    }
}

void launch_trace_monitoring_wrapper(
    const void* d_synapses,
    int num_synapses,
    void* d_trace_stats
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    trace_monitoring_kernel<<<grid, block>>>(
        static_cast<const GPUSynapse*>(d_synapses),
        num_synapses,
        static_cast<float*>(d_trace_stats));
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in trace_monitoring: %s\n", hipGetErrorString(error));
    }
}

void launch_reward_modulation_wrapper(
    void* d_synapses,
    void* d_neurons,
    float reward,
    float current_time,
    float dt,
    int num_synapses
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    reward_modulation_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<GPUNeuronState*>(d_neurons),
        reward, current_time, dt, num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in reward_modulation: %s\n", hipGetErrorString(error));
    }
}

void launch_hebbian_learning_wrapper(
    void* d_synapses,
    const void* d_neurons,
    float current_time,
    float dt,
    int num_synapses
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    hebbian_learning_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<const GPUNeuronState*>(d_neurons),
        current_time, dt, num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in hebbian_learning: %s\n", hipGetErrorString(error));
    }
}

void launch_bcm_learning_wrapper(
    void* d_synapses,
    const void* d_neurons,
    float learning_rate,
    float dt,
    int num_synapses
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    bcm_learning_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<const GPUNeuronState*>(d_neurons),
        learning_rate, dt, num_synapses);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in bcm_learning: %s\n", hipGetErrorString(error));
    }
}

void launch_correlation_learning_wrapper(
    void* d_synapses,
    const void* d_neurons,
    void* d_correlation_matrix,
    float learning_rate,
    float dt,
    int num_synapses,
    int matrix_size
) {
    dim3 block(256);
    dim3 grid((num_synapses + block.x - 1) / block.x);
    
    correlation_learning_kernel<<<grid, block>>>(
        static_cast<GPUSynapse*>(d_synapses),
        static_cast<const GPUNeuronState*>(d_neurons),
        static_cast<float*>(d_correlation_matrix),
        learning_rate, dt, num_synapses, matrix_size);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in correlation_learning: %s\n", hipGetErrorString(error));
    }
}

void launch_reward_prediction_error_wrapper(
    const void* d_actual_reward,
    void* d_predicted_rewards,
    int num_timesteps
) {
    dim3 block(256);
    dim3 grid((num_timesteps + block.x - 1) / block.x);
    
    reward_prediction_error_kernel<<<grid, block>>>(
        static_cast<const float*>(d_actual_reward),
        static_cast<float*>(d_predicted_rewards),
        num_timesteps);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in reward_prediction_error: %s\n", hipGetErrorString(error));
    }
}

} // extern "C"