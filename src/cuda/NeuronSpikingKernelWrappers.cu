// ============================================================================
// CUDA KERNEL WRAPPER FUNCTIONS FOR NEUROSPIKINGKERNELS
// File: src/cuda/NeuronSpikingKernelWrappers.cu
// ============================================================================

#include "NeuroGen/cuda/NeuronSpikingKernels.cuh"
#include "NeuroGen/cuda/GPUNeuralStructures.h"
#include "NeuroGen/cuda/NeuronModelConstants.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <chrono>

// ============================================================================
// EXTERNAL LINKAGE WRAPPER FUNCTIONS
// ============================================================================

/**
 * @brief Host wrapper for updateNeuronSpikes kernel with correct signature
 * 
 * This wrapper ensures proper linkage for the NetworkCUDA class while
 * maintaining the biologically accurate spike detection implementation.
 */
extern "C" void launchUpdateNeuronSpikesHost(GPUNeuronState* neurons,
                                             int num_neurons,
                                             float current_time,
                                             float dt) {
    dim3 block(256);
    dim3 grid((num_neurons + block.x - 1) / block.x);

    // Launch the actual CUDA kernel with proper parameters
    updateNeuronSpikes<<<grid, block>>>(neurons, num_neurons, current_time, dt);
    
    // Check for kernel launch errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error in launchUpdateNeuronSpikesHost: %s\n",
                hipGetErrorString(error));
    }
    
    // Synchronize to ensure completion
    hipDeviceSynchronize();
}

/**
 * @brief Host wrapper for countSpikesKernel with correct signature
 * 
 * This wrapper provides the exact function signature expected by NetworkCUDA
 * while leveraging our advanced spike counting implementation.
 */
extern "C" void launchCountSpikes(const GPUNeuronState* neurons,
                                   int* spike_count,
                                   int num_neurons,
                                   float current_time) {
    dim3 block(256);
    dim3 grid((num_neurons + block.x - 1) / block.x);

    // Launch the actual CUDA kernel
    countSpikesKernel<<<grid, block>>>(neurons, spike_count, num_neurons, current_time);
    
    // Check for kernel launch errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error in launchCountSpikes: %s\n",
                hipGetErrorString(error));
    }
    
    // Synchronize to ensure completion
    hipDeviceSynchronize();
}

/**
 * @brief Advanced wrapper for comprehensive spike processing
 * 
 * This wrapper provides a unified interface for all spike-related processing,
 * enabling the breakthrough neural architecture to handle complex spike dynamics
 * with optimal performance.
 */
extern "C" void processNeuralSpikes(GPUNeuronState* neurons, int* spike_count,
                                   float current_time,
                                   int num_neurons, float dt) {
    if (!neurons || !spike_count || num_neurons <= 0) {
        printf("Error: Invalid parameters for processNeuralSpikes\n");
        return;
    }
    
    dim3 block(256);
    dim3 grid((num_neurons + block.x - 1) / block.x);
    
    // Reset spike counter
    hipMemset(spike_count, 0, sizeof(int));
    
    // Step 1: Update neuron spike states with biological realism
    updateNeuronSpikes<<<grid, block>>>(neurons, num_neurons, current_time, dt);
    hipDeviceSynchronize();
    
    // Step 2: Count spikes for network statistics
    countSpikesKernel<<<grid, block>>>(neurons, spike_count, num_neurons, current_time);
    hipDeviceSynchronize();
    
    // Check for any errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in processNeuralSpikes: %s\n", hipGetErrorString(error));
    }
}

/**
 * @brief Wrapper for modular spike processing with attention mechanisms
 * 
 * This advanced wrapper supports the modular neural architecture by providing
 * module-aware spike processing with attention-based modulation.
 */
extern "C" void processModularSpikes(GPUNeuronState* neurons, int* spike_count,
                                    int* module_assignments, float* attention_weights,
                                    float current_time,
                                    int num_neurons, int num_modules, float dt) {
    if (!neurons || !spike_count || num_neurons <= 0) {
        return;
    }
    
    dim3 block(256);
    dim3 grid((num_neurons + block.x - 1) / block.x);
    
    // Reset spike counter
    hipMemset(spike_count, 0, sizeof(int));
    
    // Process spikes with modular awareness
    if (module_assignments && attention_weights) {
        // Use advanced modular spike processing
        launchProcessModularInteractions(neurons, num_neurons, module_assignments,
                                       attention_weights, nullptr, current_time);
    }
    
    // Standard spike detection and counting
    updateNeuronSpikes<<<grid, block>>>(neurons, num_neurons, current_time, dt);
    countSpikesKernel<<<grid, block>>>(neurons, spike_count, num_neurons, current_time);
    
    hipDeviceSynchronize();
}

// ============================================================================
// COMPATIBILITY LAYER FOR LEGACY INTERFACES
// ============================================================================

/**
 * @brief Legacy compatibility wrapper for older NetworkCUDA interfaces
 */
extern "C" void launchSpikeDetection(GPUNeuronState* d_neurons, int* d_spike_count,
                                    int num_neurons, float current_time) {
    processNeuralSpikes(d_neurons, d_spike_count, current_time, num_neurons, 0.1f);
}

/**
 * @brief Simplified interface for basic spike counting
 */
extern "C" int countActiveNeurons(const GPUNeuronState* neurons, int num_neurons, float current_time) {
    if (!neurons || num_neurons <= 0) return 0;
    
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));
    
    countSpikesKernel<<<(num_neurons + 255)/256, 256>>>(neurons, d_count, num_neurons, current_time);
    hipDeviceSynchronize();
    
    int h_count = 0;
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);

    return h_count;
}

// Forward declaration of advanced modular interaction helper implemented in
// NeuronSpikingKernels.cu
extern "C" void launchProcessModularInteractions(GPUNeuronState* neurons, int num_neurons,
                                                int* module_assignments, float* attention_weights,
                                                float* global_inhibition, float current_time);

// ============================================================================
// PERFORMANCE MONITORING FUNCTIONS
// ============================================================================

/**
 * @brief Monitor spike processing performance for optimization
 */
extern "C" float benchmarkSpikeProcessing(GPUNeuronState* neurons, int num_neurons,
                                         int iterations = 100) {
    if (!neurons || num_neurons <= 0) return 0.0f;
    
    int* d_spike_count;
    hipMalloc(&d_spike_count, sizeof(int));
    
    // Warm up
    for (int i = 0; i < 10; i++) {
        updateNeuronSpikes<<<(num_neurons + 255)/256, 256>>>(neurons, num_neurons, 0.0f, 1.0f);
    }
    hipDeviceSynchronize();
    
    // Benchmark
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < iterations; i++) {
        updateNeuronSpikes<<<(num_neurons + 255)/256, 256>>>(neurons, num_neurons, 0.0f, 1.0f);
        countSpikesKernel<<<(num_neurons + 255)/256, 256>>>(neurons, d_spike_count, num_neurons, 0.0f);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    
    hipFree(d_spike_count);
    
    return static_cast<float>(duration.count()) / iterations; // microseconds per iteration
}

/**
 * @brief Validate spike processing accuracy for debugging
 */
extern "C" bool validateSpikeProcessing(GPUNeuronState* neurons, int num_neurons) {
    if (!neurons || num_neurons <= 0) return false;
    
    // Copy neurons to host for validation
    std::vector<GPUNeuronState> h_neurons(num_neurons);
    hipMemcpy(h_neurons.data(), neurons, num_neurons * sizeof(GPUNeuronState), 
               hipMemcpyDeviceToHost);
    
    // Basic validation checks
    bool valid = true;
    for (int i = 0; i < num_neurons; i++) {
        const auto& neuron = h_neurons[i];
        
        // Check for reasonable voltage values
        if (neuron.V < -100.0f || neuron.V > 100.0f) {
            printf("Warning: Neuron %d has unreasonable voltage: %f\n", i, neuron.V);
            valid = false;
        }
        
        // Check for reasonable calcium values
        for (int c = 0; c < 4; c++) {
            if (neuron.ca_conc[c] < 0.0f || neuron.ca_conc[c] > 50.0f) {
                printf("Warning: Neuron %d compartment %d has unreasonable calcium: %f\n", 
                       i, c, neuron.ca_conc[c]);
                valid = false;
            }
        }
    }
    
    return valid;
}
