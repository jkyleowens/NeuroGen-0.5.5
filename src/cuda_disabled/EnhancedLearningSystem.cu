#include "hip/hip_runtime.h"
// ============================================================================
// ENHANCED LEARNING SYSTEM CUDA IMPLEMENTATION
// File: src/cuda/EnhancedLearningSystem.cu
// ============================================================================

#include "NeuroGen/EnhancedLearningSystem.h"
#include "NeuroGen/cuda/GPUNeuralStructures.h"
#include "NeuroGen/cuda/EnhancedSTDPKernel.cuh"
#include "NeuroGen/cuda/HebbianLearningKernel.cuh"
#include "NeuroGen/cuda/HomeostaticMechanismsKernel.cuh"
#include "NeuroGen/cuda/EligibilityAndRewardKernels.cuh"
#include "NeuroGen/cuda/RewardModulationKernel.cuh"
#include "NeuroGen/cuda/NeuromodulationKernels.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

// ============================================================================
// CUDA IMPLEMENTATION OF EnhancedLearningSystem GPU METHODS
// These methods handle GPU-specific learning operations
// ============================================================================

void EnhancedLearningSystem::updateLearningGPU(GPUSynapse* synapses, 
                                              GPUNeuronState* neurons,
                                              float current_time, 
                                              float dt,
                                              float external_reward) {
    
    if (!cuda_initialized_) {
        return;
    }
    
    // Store device pointers (cast to void* for header compatibility)
    d_synapses_ptr_ = static_cast<void*>(synapses);
    d_neurons_ptr_ = static_cast<void*>(neurons);
    
    // Update main learning mechanisms
    update_learning(current_time, dt, external_reward);
    
    // Synchronize GPU execution
    hipStreamSynchronize(learning_stream_);
}

void EnhancedLearningSystem::resetEpisodeGPU(bool reset_traces, bool reset_rewards) {
    if (!cuda_initialized_) {
        return;
    }
    
    if (reset_traces) {
        // Reset eligibility traces
        launch_eligibility_reset_gpu();
    }
    
    if (reset_rewards) {
        // Reset reward-related variables
        float zero_values[4] = {0.0f, 0.0f, 0.0f, 0.0f};
        if (d_reward_signals_ptr_) {
            hipMemcpy(d_reward_signals_ptr_, zero_values, 4 * sizeof(float), hipMemcpyHostToDevice);
        }
    }
    
    hipStreamSynchronize(learning_stream_);
}

EnhancedLearningSystem::LearningStats EnhancedLearningSystem::getStatisticsGPU() const {
    LearningStats stats;
    
    // Initialize with current tracked values
    stats.total_weight_change = total_weight_change_;
    stats.average_trace_activity = average_eligibility_trace_;
    stats.current_dopamine_level = baseline_dopamine_;
    stats.prediction_error = 0.0f;
    stats.network_activity = 0.0f;
    stats.plasticity_updates = 0;
    
    // Get network activity from GPU if available
    if (cuda_initialized_ && d_trace_stats_ptr_) {
        float trace_stats[4] = {0};
        hipMemcpy(trace_stats, d_trace_stats_ptr_, 4 * sizeof(float), hipMemcpyDeviceToHost);
        stats.network_activity = trace_stats[0];
        stats.average_trace_activity = trace_stats[1];
    }
    
    return stats;
}

// ============================================================================
// PRIVATE CUDA HELPER METHODS
// ============================================================================

void EnhancedLearningSystem::launch_eligibility_reset_gpu() {
    if (!cuda_initialized_ || !d_synapses_ptr_) {
        return;
    }
    
    // Launch eligibility reset kernel
    GPUSynapse* synapses = static_cast<GPUSynapse*>(d_synapses_ptr_);
    
    dim3 blockSize(256);
    dim3 gridSize((num_synapses_ + blockSize.x - 1) / blockSize.x);
    
    // Reset eligibility traces to zero
    hipMemset(synapses, 0, num_synapses_ * sizeof(float)); // Reset eligibility field
    
    hipStreamSynchronize(learning_stream_);
}

void EnhancedLearningSystem::reset_eligibility_traces_gpu() {
    if (!cuda_initialized_ || !d_trace_stats_ptr_) {
        return;
    }
    
    // Reset eligibility traces using memset (simple approach)
    size_t synapse_memory_size = num_synapses_ * sizeof(float) * 4; // Estimate for eligibility traces
    hipMemset(d_synapses_ptr_, 0, synapse_memory_size);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error in eligibility reset: %s\n", hipGetErrorString(error));
    }
}

// ============================================================================
// CUDA KERNEL LAUNCH HELPERS
// ============================================================================

void EnhancedLearningSystem::update_performance_metrics_gpu() {
    if (!cuda_initialized_ || !d_trace_stats_ptr_) {
        return;
    }
    
    // Copy trace statistics from GPU
    float trace_stats[4];
    hipMemcpy(trace_stats, d_trace_stats_ptr_, 4 * sizeof(float), hipMemcpyDeviceToHost);
    
    // Update performance metrics
    average_eligibility_trace_ = trace_stats[0] / std::max(1, num_synapses_);
    
    // Update learning progress based on weight changes
    float progress = std::min(1.0f, total_weight_change_ / (num_synapses_ * 0.1f));
    learning_progress_ = progress;
    
    // Update total weight change
    total_weight_change_ += trace_stats[2]; // Assuming trace_stats[2] contains weight change magnitude
}

// ============================================================================
// CUDA ERROR CHECKING
// ============================================================================

void checkCudaErrors() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in Enhanced Learning System: %s\n", 
               hipGetErrorString(error));
    }
}
